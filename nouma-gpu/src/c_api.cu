#include "hip/hip_runtime.h"
#include "phantom.h"

#include <vector>
#include <algorithm>

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

template<typename T>
inline void print_vector(std::vector<T> vec, std::size_t print_size = 4, int prec = 3) {
    /*
    Save the formatting information for std::cout.
    */
    std::ios old_fmt(nullptr);
    old_fmt.copyfmt(std::cout);

    std::size_t slot_count = vec.size();

    std::cout << std::fixed << std::setprecision(prec);
    std::cout << std::endl;
    if (slot_count <= 2 * print_size) {
        std::cout << "    [";
        for (std::size_t i = 0; i < slot_count; i++) {
            std::cout << " " << vec[i] << ((i != slot_count - 1) ? "," : " ]\n");
        }
    }
    else {
        vec.resize(std::max(vec.size(), 2 * print_size));
        std::cout << "    [";
        for (std::size_t i = 0; i < print_size; i++) {
            std::cout << " " << vec[i] << ",";
        }
        if (vec.size() > 2 * print_size) {
            std::cout << " ...,";
        }
        for (std::size_t i = slot_count - print_size; i < slot_count; i++) {
            std::cout << " " << vec[i] << ((i != slot_count - 1) ? "," : " ]\n");
        }
    }
    std::cout << std::endl;

    /*
    Restore the old std::cout formatting.
    */
    std::cout.copyfmt(old_fmt);
}

extern "C" {

EncryptionParameters *EncryptionParameters_CKKSCreate(size_t poly_modulus_degree, const int *bit_sizes, size_t length) {
    EncryptionParameters *params =  new EncryptionParameters(scheme_type::ckks);
    params->set_poly_modulus_degree(poly_modulus_degree);
    params->set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, vector<int>(bit_sizes, bit_sizes + length)));
    return params;
}

void EncryptionParameters_Delete(EncryptionParameters *params) {
    delete params;
}


PhantomContext *Context_New(EncryptionParameters *params) {
    return new PhantomContext(*params);
}

void Context_Delete(PhantomContext *ctx) {
    delete ctx;
}

PhantomCKKSEncoder *CKKSEncoder_New(PhantomContext *ctx) {
    return new PhantomCKKSEncoder(*ctx);
}

void CKKSEncoder_Delete(PhantomCKKSEncoder *encoder) {
    delete encoder;
}

PhantomSecretKey *SecretKey_Create(PhantomContext *ctx) {
    auto secret_key = new PhantomSecretKey(*ctx);
    return secret_key;
}

PhantomPublicKey *PublicKey_Create(PhantomContext *ctx, PhantomSecretKey *secret_key) {
    return secret_key->gen_publickey(ctx);
}

PhantomRelinKey *RelinKey_Create(PhantomContext *ctx, PhantomSecretKey *secret_key) {
    return secret_key->gen_relinkey(ctx);
}

PhantomGaloisKey *GaloisKey_Create(PhantomContext *ctx, PhantomSecretKey *secret_key) {
    return secret_key->create_galois_keys(ctx);
}

void SecretKey_Delete(PhantomSecretKey *secret_key) {
    delete secret_key;
}


void PublicKey_Delete(PhantomPublicKey *public_key) {
    delete public_key;
}


void RelinKey_Delete(PhantomRelinKey *relin_key) {
    delete relin_key;
}

void GaloisKey_Delete(PhantomGaloisKey *galois_key) {
    delete galois_key;
}

PhantomPlaintext *Plaintext_New() {
    return new PhantomPlaintext();
}

PhantomCiphertext *Ciphertext_New() {
    return new PhantomCiphertext();
}

void Plaintext_Delete(PhantomPlaintext *plaintext) {
    delete plaintext;
}


void Ciphertext_Delete(PhantomCiphertext *ciphertext) {
    delete ciphertext;
}


void CKKSEncoder_Encode(PhantomCKKSEncoder *encoder, PhantomContext *ctx, double *inputs, size_t size, double scale, PhantomPlaintext *plaintext) {
    auto slot_count = encoder->slot_count();
    vector<double> vec(size, 0);
    vec.assign(inputs, inputs + min(slot_count, size));
    vec.resize(slot_count);
    fill(vec.begin() + min(slot_count, size), vec.end(), 0);
    encoder->encode(*ctx, vec, scale, *plaintext);
}

void CKKSEncoder_Decode(PhantomCKKSEncoder *encoder, PhantomContext *ctx, PhantomPlaintext *plaintext, double *outputs, size_t size) {
    auto slot_count = encoder->slot_count();
    vector<double> vec;
    encoder->decode(*ctx, *plaintext, vec);
    copy(vec.begin(), vec.begin() + min(slot_count, size), outputs);
}

void SecretKey_EncryptSymmetric(PhantomSecretKey *secret_key, PhantomContext *ctx, PhantomPlaintext *plaintext, PhantomCiphertext *ciphertext) {
    secret_key->encrypt_symmetric(*ctx, *plaintext, *ciphertext);
}

void PublicKey_EncryptAsymmetric(PhantomPublicKey *public_key, PhantomContext *ctx, PhantomPlaintext *plaintext, PhantomCiphertext *ciphertext) {
    public_key->encrypt_asymmetric(*ctx, *plaintext, *ciphertext);
}

void SecretKey_Decrypt(PhantomSecretKey *secret_key, PhantomContext *ctx, PhantomCiphertext *ciphertext, PhantomPlaintext *plaintext) {
    secret_key->decrypt(*ctx, *ciphertext, *plaintext);
}

void Dot_Product(PhantomContext *ctx, PhantomCiphertext *src1, PhantomCiphertext *src2, PhantomCiphertext *dst, PhantomRelinKey *relin_keys, PhantomGaloisKey *galois_keys) {
    *dst = *src1;
    multiply_inplace(*ctx, *dst, *src2);

    relinearize_inplace(*ctx, *dst, *relin_keys);

    PhantomCiphertext r;
    for (int i = 0; i < 12; i++) {
        r = *dst;
        rotate_vector_inplace(*ctx, r, pow(2, i), *galois_keys);
        add_inplace(*ctx, *dst, r);
    }

}

} // extern "C"