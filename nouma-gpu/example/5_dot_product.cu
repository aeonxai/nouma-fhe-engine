#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "example.h"
#include "phantom.h"
#include "util.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

void example_dot_product() {
    cout << "Example: Dot Product" << endl;
    EncryptionParameters parms(scheme_type::ckks);
    size_t poly_modulus_degree = 8192;
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(poly_modulus_degree, { 60, 40, 40, 60 }));
    PhantomContext context(parms);
    PhantomSecretKey secret_key(context);
    PhantomGaloisKey galois_keys = secret_key.create_galois_keys(context);
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);



    PhantomCKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();
    cout << "Number of slots: " << slot_count << endl;

    double scale = pow(2.0, 40);

    // vector<hipDoubleComplex> v1, v2;

    // v1.push_back(make_hipDoubleComplex(1, 0));
    // v1.push_back(make_hipDoubleComplex(2, 0));
    // v1.push_back(make_hipDoubleComplex(3, 0));
    // v1.push_back(make_hipDoubleComplex(4, 0));

    // v2.push_back(make_hipDoubleComplex(5, 0));
    // v2.push_back(make_hipDoubleComplex(6, 0));
    // v2.push_back(make_hipDoubleComplex(7, 0));
    // v2.push_back(make_hipDoubleComplex(8, 0));

    // for (int i = 5; i <= slot_count; i++) {
    //     v1.push_back(make_hipDoubleComplex(0, 0));
    //     v2.push_back(make_hipDoubleComplex(0, 0));
    // }

    vector<double> v1, v2;

    v1.push_back(1);
    v1.push_back(2);
    v1.push_back(3);
    v1.push_back(4);

    v2.push_back(5);
    v2.push_back(6);
    v2.push_back(7);
    v2.push_back(8);

    for (int i = 5; i <= slot_count; i++) {
        v1.push_back(0);
        v2.push_back(0);
    }
    

    PhantomPlaintext p1, p2;
    encoder.encode(context, v1, scale, p1);
    encoder.encode(context, v2, scale, p2);

    PhantomCiphertext c1, c2;
    secret_key.encrypt_symmetric(context, p1, c1);
    secret_key.encrypt_symmetric(context, p2, c2);


    PhantomCiphertext c = c1;
    multiply_inplace(context, c, c2);

    relinearize_inplace(context, c, relin_keys);

    PhantomCiphertext r;
    for (int i = 0; i < 12; i++) {
        r = c;
        rotate_vector_inplace(context, r, pow(2, i), galois_keys);
        add_inplace(context, c, r);
    }

    PhantomPlaintext p;
    secret_key.decrypt(context, c, p);

    vector<double> v;
    encoder.decode(context, p, v);

    print_vector(v1, 3, 7);
    print_vector(v2, 3, 7);
    print_vector(v, 3, 7);

    // vector<double> v3(5, 0);
    // print_vector(v3, 3, 7);

}